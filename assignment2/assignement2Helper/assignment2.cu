#include "hip/hip_runtime.h"
// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define filter_width 15
#define filter_height 15
#define TILE_WIDTH 32 + 2*(filter_width/2)

// Define the files that are to be save and the reference images for validation

const char *imageFilename = "image21.pgm";

const char *sampleName = "simpleTexture";
int option = 4;


////////////////////////////////////////////////////////////////////////////////
// Constants
// Texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

//declare constant memory
__constant__ float d_filter_constant[filter_width*filter_height];

// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
//! Convolute image
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////

void cpuConvolution(float *image,
                    float *outImage,
                    float *filter,
                    int width,
                    int height,
                    int pad_w,
                    int pad_h)
{
    //time vars
    clock_t start, end;
    start = clock();
    //calculate the convolution per image pixel
    for(int i = 0; i < height; i++){
        for(int j = 0; j < width; j++){
            double convolution_value = 0;
            for(int k = 0; k < filter_height; k++){
                for(int l = 0; l < filter_width; l++){
                    convolution_value += filter[l+k*filter_width]*image[((j+l)+(i+k)*(width+(2*pad_w)))];
                }
            }
            // write pixel to output array
            outImage[j + i*width] = convolution_value;
        }
    }
    end = clock();
    printf("Processing time: %f (ms)\n", (double)((end-start)/1000));
}

__global__ void naiveConvolutionKernel(float *image,
                                      float *outImage,
                                      float *filter,
                                      int width,
                                      int pad_w)
{
    // calculate coordinates
    int i = threadIdx.y + blockIdx.y*blockDim.y;
    int j = threadIdx.x + blockIdx.x*blockDim.x;

    //calculate the convolution per image pixel
    double convolution_value = 0;
    for(int k = 0; k < filter_height; k++){
        for(int l = 0; l < filter_width; l++){
            convolution_value += filter[l+k*filter_width]*image[((j+l)+(i+k)*(width+(2*pad_w)))];
        }
    }

    // save value to output data
    outImage[j + i*width] = convolution_value;
}

__global__ void sharedConvolutionKernel(float *image,
                                      float *outImage,
                                      float *filter,
                                      int width,
                                      int pad_w)
{
    //allocate shared memory space
    __shared__ float s_image[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_filter[filter_height][filter_width];

    // calculate the row & column index of the element
    int blockx = blockIdx.x;
    int blocky = blockIdx.y;
    int threadx = threadIdx.x;
    int thready = threadIdx.y;
    int i = blocky*(TILE_WIDTH - (2*pad_w)) + thready;
    int j = blockx*(TILE_WIDTH - (2*pad_w))  + threadx;

    // load data into shared memory

    // load filter into shared memory in 1 thread per block along with the image tile we wish to use for the convolution of this block

    for(int k = 0; k < filter_height; k++){
        for(int l = 0; l < filter_width; l++){
			if(threadx == 0 && thready == 0){
				s_filter[k][l] = filter[l+k*filter_width];
			}
			s_image[thready+k][threadx+l] = image[(j+l)+((i+k)*(width+(2*pad_w)))];
        }
    }
    __syncthreads();


    //calculate the convolution per image pixel using shared memory
    double convolution_value = 0;
    for(int k = 0; k < filter_height; k++){
        for(int l = 0; l < filter_width; l++){
            convolution_value += s_filter[k][l]*s_image[thready+k][threadx+l];
        }
    }

    // save value to output data
    outImage[j + i*width] = convolution_value;
}

__global__ void constantSharedConvolutionKernel(float *image,
                                      float *outImage,
                                      int width,
                                      int pad_w)
{
    //allocate shared memory space
    __shared__ float s_image[TILE_WIDTH][TILE_WIDTH];

    // calculate the row & column index of the element
    int blockx = blockIdx.x;
    int blocky = blockIdx.y;
    int threadx = threadIdx.x;
    int thready = threadIdx.y;
    int i = blocky*(TILE_WIDTH - (2*pad_w)) + thready;
    int j = blockx*(TILE_WIDTH - (2*pad_w))  + threadx;

    // load data into shared memory

    //  load the image tile we wish to use for the convolution of this block into shared memory

    for(int k = 0; k < filter_height; k++){
        for(int l = 0; l < filter_width; l++){
			s_image[thready+k][threadx+l] = image[(j+l)+((i+k)*(width+(2*pad_w)))];
        }
    }
    __syncthreads();

    //calculate the convolution per image pixel using shared memory
    double convolution_value = 0;
    for(int k = 0; k < filter_height; k++){
        for(int l = 0; l < filter_width; l++){
            convolution_value += d_filter_constant[l+k*filter_width]*s_image[thready+k][threadx+l];
        }
    }

    // save value to output data
    outImage[j + i*width] = convolution_value;
}

__global__ void textureConvolutionKernel(float *outImage,
                                      int width,
                                      int height,
                                      int img_width)
{
    // calculate coordinates
    int i = threadIdx.y + blockIdx.y*blockDim.y;
    int j = threadIdx.x + blockIdx.x*blockDim.x;

    //calculate the convolution per image pixel
    double convolution_value = 0;
    for(int k = 0; k < filter_height; k++){
        for(int l = 0; l < filter_width; l++){
            convolution_value += d_filter_constant[l+k*filter_width]*tex2D(tex, j - height + k + 0.5f, i - width + l + 0.5f);
        }
    }

    // save value to output data
    outImage[j + (i*(img_width))] = convolution_value;
}
////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

void filter_gen(float* filter, int width, int height);

void padded_populate(float* padded, float* image, int width, int height/*, float *filter*/);

void executeNaiveConvolution(int height, int width, float *d_padded_image, float *dData, float *d_filter, int padWidth);

void executeSharedConvolution(int height, int width, float *d_padded_image, float *dData, float *d_filter, int padWidth);

void executeConstantSharedConvolution(int height, int width, float *d_padded_image, float *dData, int padWidth);

void executeTextureConvolution(int height, int width, float *dData, int padHeight, int padWidth);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    //printf("%s starting...\n", sampleName);

    // Process command-line arguments
    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **) argv, "input"))
        {
            getCmdLineArgumentString(argc,
                                     (const char **) argv,
                                     "input",
                                     (char **) &imageFilename);
        }
    }
    for(int i = 0; i<=4; i++){
        option = i;
        runTest(argc, argv);
    }


    hipDeviceReset();
    //printf("%s completed, returned %s\n",
    //       sampleName,
    //       testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv)
{
    //method name
    char * name;
    //int devID = findCudaDevice(argc, (const char **) argv);
    // load image from disk
    float *hData = NULL;
    //host filter
    float* h_filter = NULL;
    //width and height of original image
    unsigned int width, height;
    char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

    if (imagePath == NULL)
    {
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }
    // load image into hData
    sdkLoadPGM(imagePath, &hData, &width, &height);

    ////////////////////////////////////
    // generate filter
    ////////////////////////////////////

    h_filter = (float *)malloc(filter_width*filter_height*sizeof(float));
    filter_gen(h_filter,filter_width,filter_height);

    //total size of image
    unsigned int size = width * height * sizeof(float);
    // Allocate mem for the result on host side
    float *hOutputData = (float *) malloc(size);

    //printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

    //option 0 is texture
    if(option == 0){

        // array for output image
        float *dData = NULL;
        checkCudaErrors(hipMalloc((void **) &dData, size));

        // Allocate array and copy image data
        hipChannelFormatDesc channelDesc =
            hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
        hipArray *cuArray;
        checkCudaErrors(hipMallocArray(&cuArray,
                                        &channelDesc,
                                        width,
                                        height));
        checkCudaErrors(hipMemcpyToArray(cuArray,
                                          0,
                                          0,
                                          hData,
                                          size,
                                          hipMemcpyHostToDevice));

        // Set texture parameters
        tex.addressMode[0] = hipAddressModeBorder;
        tex.addressMode[1] = hipAddressModeBorder;
        tex.filterMode = hipFilterModeLinear;

        // Amount of padding
        int padWidth = (int)2*floor(filter_width/2);
        int padHeight = (int)2*floor(filter_height/2);

        // Bind the array to the texture
        checkCudaErrors(hipBindTextureToArray(tex, cuArray, channelDesc));

        //copy host filter data to constant memory
        hipMemcpyToSymbol(HIP_SYMBOL(d_filter_constant), h_filter, sizeof(float)*filter_width*filter_height);

        printf("Texture ");

        executeTextureConvolution(height, width, dData, padHeight, padWidth);

        // copy result from device to host
        checkCudaErrors(hipMemcpy(hOutputData,
                                   dData,
                                   size,
                                   hipMemcpyDeviceToHost));

        // free memory
        checkCudaErrors(hipFree(dData));
        checkCudaErrors(hipFreeArray(cuArray));
        name = (char *)"Texture";

    }else if(option >= 1){

        // padded image
        float *hDataPadded = NULL;

        // Amount of padding
        int padWidth = (int)2*floor(filter_width/2);
        int padHeight = (int)2*floor(filter_height/2);

        // Allocate memory for padded images
        hDataPadded = (float *)malloc(sizeof(float)*(width+(padWidth))*(height+(padHeight)));


        // populate padded image with hData
        padded_populate(hDataPadded, hData, width, height/*, h_filter*/);

        if(option>1){

            //this is gpu related kernel initialization

            // Allocate device memory for result
            float *dData = NULL;
            checkCudaErrors(hipMalloc((void **) &dData, size));

            // Allocate device memory for padded image
            float* d_padded_image = NULL;
            checkCudaErrors(hipMalloc((void **)&d_padded_image, sizeof(float)*(width+(padWidth))*(height+(padHeight))));
            checkCudaErrors(hipMemcpy(d_padded_image, hDataPadded, sizeof(float)*(width+(padWidth))*(height+(padHeight)), hipMemcpyHostToDevice));


            if(option == 2){
                // Allocate device memory for filter
                float* d_filter = NULL;
                checkCudaErrors(hipMalloc((void **)&d_filter, sizeof(float)*filter_width*filter_height));
                checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float)*filter_width*filter_height, hipMemcpyHostToDevice));
                printf("Naive ");
                //execute chosen kernel
                executeNaiveConvolution(height, width, d_padded_image, dData, d_filter, padWidth);
                //printf("Exectuted naive convolution \n");
                name = (char *)"Naive";
                //free memory
                checkCudaErrors(hipFree(d_filter));
            }else if(option == 3){
                float* d_filter = NULL;
                checkCudaErrors(hipMalloc((void **)&d_filter, sizeof(float)*filter_width*filter_height));
                checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float)*filter_width*filter_height, hipMemcpyHostToDevice));
                printf("Shared ");
                //execute chosen kernel
                executeSharedConvolution(height, width, d_padded_image, dData, d_filter, padWidth);
                //printf("Exectuted shared convolution \n");
                name = (char *)"Shared";
                //free memory
                checkCudaErrors(hipFree(d_filter));
            }else if(option == 4){

                //copy host filter data to constant memory
                hipMemcpyToSymbol(HIP_SYMBOL(d_filter_constant), h_filter, sizeof(float)*filter_width*filter_height);
                printf("Constant Shared ");
                //execute chosen kernel
                executeConstantSharedConvolution(height, width, d_padded_image, dData, padWidth);
                name = (char *)"Constant_Shared";
                //printf("Exectuted constant shared convolution \n");
            }

            // copy result from device to host
            checkCudaErrors(hipMemcpy(hOutputData,
                                       dData,
                                       size,
                                       hipMemcpyDeviceToHost));

            // free memory
            checkCudaErrors(hipFree(dData));
            checkCudaErrors(hipFree(d_padded_image));
        }else if(option == 1){
            //cpu convolution
            printf("CPU ");
            cpuConvolution(hDataPadded, hOutputData, h_filter, width, height, padWidth/2, padHeight/2);
            name = (char *)"CPU";
            //printf("Exectuted cpu convolution \n");
        }else{
            //print ERROR
            perror("Invalid option");
        }

        //free memory
        free(hDataPadded);

    }else{
        //print ERROR
        perror("Invalid option");
    }

    // Write result to file
    char outputFilename[1024];
    strcpy(outputFilename, imagePath);
    strcpy(outputFilename + strlen(imagePath) - 4, "_");
    strcpy(outputFilename + strlen(imagePath) - 4 + 1, name);
    strcpy(outputFilename + strlen(imagePath) - 4 + 1 + strlen(name), "_out.pgm");
    sdkSavePGM(outputFilename, hOutputData, width, height);
    //printf("Wrote '%s'\n", outputFilename);

    //free space
    free(imagePath);
    free(h_filter);
    free(hData);
    free(hOutputData);
}

// generate filter for image
void filter_gen(float* filter, int width, int height){
  for(int i = 0; i < width*height; i++){
    filter[i] = (float)1/(width*height);
  }
  //filter[(int)(floor(width/2) + floor(height/2)*width)] = (float)(-1)*(4/(width*height));
}

// populate padded images
void padded_populate(float* padded, float* image, int width, int height/*, float *filter*/){
    // padded border increase
    int w = (int)floor(filter_width/2);
    int h = (int)floor(filter_height/2);

    // initialize padded array to 0
    for(int i = 0; i<(height+(2*h));i++){
      for(int j = 0; j<(width+(2*w)); j++){
        padded[j+(i*(width+(2*w)))] = (float)0;
      }
    }
    //populate padded array with image
    for(int i = 0; i<height;i++){
      for(int j = 0; j<width; j++){;
          padded[(j+w)+(i+h)*(width+2*w)] = image[j+(i*width)];
      }
    }
}

void executeNaiveConvolution(int height, int width, float *d_padded_image, float *dData, float *d_filter, int padWidth){

    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    // Execute the kernel warmup
    naiveConvolutionKernel<<<dimGrid, dimBlock, 0>>>(d_padded_image, dData, d_filter, width, padWidth/2);

    checkCudaErrors(hipDeviceSynchronize());

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Execute the kernel
    naiveConvolutionKernel<<<dimGrid, dimBlock, 0>>>(d_padded_image, dData, d_filter, width, padWidth/2);

    // Check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    //printf("%.2f Mpixels/sec\n",
           //(width *height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    sdkDeleteTimer(&timer);

}

void executeSharedConvolution(int height, int width, float *d_padded_image, float *dData, float *d_filter, int padWidth){

    dim3 dimBlock((TILE_WIDTH - (padWidth)), (TILE_WIDTH - (padWidth)), 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    // Execute the kernel warmup
    sharedConvolutionKernel<<<dimGrid, dimBlock, 0>>>(d_padded_image, dData, d_filter, width, padWidth/2);

    checkCudaErrors(hipDeviceSynchronize());

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Execute the kernel
    sharedConvolutionKernel<<<dimGrid, dimBlock, 0>>>(d_padded_image, dData, d_filter, width, padWidth/2);

    // Check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    //printf("%.2f Mpixels/sec\n",
           //(width *height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    sdkDeleteTimer(&timer);

}

void executeConstantSharedConvolution(int height, int width, float *d_padded_image, float *dData, int padWidth){

    dim3 dimBlock((TILE_WIDTH - (padWidth)), (TILE_WIDTH - (padWidth)), 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    // Execute the kernel warmup
    constantSharedConvolutionKernel<<<dimGrid, dimBlock, 0>>>(d_padded_image, dData, width, padWidth/2);

    checkCudaErrors(hipDeviceSynchronize());

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Execute the kernel
    constantSharedConvolutionKernel<<<dimGrid, dimBlock, 0>>>(d_padded_image, dData, width, padWidth/2);

    // Check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    //printf("%.2f Mpixels/sec\n",
           //(width *height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    sdkDeleteTimer(&timer);
}

void executeTextureConvolution(int height, int width, float *dData, int padHeight, int padWidth){

    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    // Execute the kernel warmup
    textureConvolutionKernel<<<dimGrid, dimBlock, 0>>>(dData, padWidth/2, padHeight/2, width);

    checkCudaErrors(hipDeviceSynchronize());

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Execute the kernel
    textureConvolutionKernel<<<dimGrid, dimBlock, 0>>>(dData, padWidth/2, padHeight/2, width);

    // Check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    //printf("%.2f Mpixels/sec\n",
           //(width *height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    sdkDeleteTimer(&timer);

}
